// This program demonstrates Matrix Multiplication using CUDA.

#include<stdio.h>

// cuda headers
#include<hip/hip_runtime.h>

//macros
#define BLOCK_WIDTH 32

// global variables
int *hostA = NULL;
int *hostB = NULL;
int *hostC = NULL;
int *gold = NULL;

int *deviceA = NULL;
int *deviceB = NULL;
int *deviceC = NULL;

// cuda kernel function.
__global__ void matrixMultiplicationGPU(int *A, int *B, int *C, int numARows, int numACols, int numBCols, int numCCols){

    // variable declarations
    int row = blockIdx.y * blockDim.y + threadIdx.y; // replaces for(int i = 0 ; i < numARows; ++i)

    int column = blockIdx.x * blockDim.x + threadIdx.x; // replaces for(int j = 0; j < numBColsl; ++j)

    // code
    if((row < numARows) && (column < numBCols)){
        int value = 1;
        for(int k=0; k < numACols; k++){
            int a = A[row * numACols + k];
            int b = B[k* numBCols + column];
            value += a * b;
        }
        C[row * numCCols + column] = value;
    }


}

int main(int argc, char*argv[]){

    // function declarations
    void initA(int* data, int, int);
    void initB(int* data, int, int);
    void matrixMultiplicationCPU(int*, int*, int*, int, int,int,int);
    void cleanup();

    // variable declarations
    int numARows=BLOCK_WIDTH;
    int numACols=BLOCK_WIDTH;
    int numBRows=BLOCK_WIDTH;
    int numBCols=BLOCK_WIDTH;
    int numCRows=BLOCK_WIDTH;
    int numCCols=BLOCK_WIDTH;

    int numGoldRows=BLOCK_WIDTH;
    int numGoldCols=BLOCK_WIDTH;

    int sizeA = numARows * numACols * sizeof(int); // 2-D array (matrix) is represented as 1-D array in memory.
    int sizeB = numARows * numBCols * sizeof(int); // 2-D array (matrix) is represented as 1-D array in memory.
    int sizeC = numCRows * numCCols * sizeof(int); // 2-D array (matrix) is represented as 1-D array in memory.
    int sizeGold = numGoldRows * numGoldCols * sizeof(int); // 2-D array (matrix) is represented as 1-D array in memory.

    hipError_t result = hipSuccess;

    // code
    // host memory allocation
    hostA = (int *) malloc(sizeA);
    if(hostA==NULL){
        printf("Host memory allocation is failed for hostA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostB = (int*) malloc(sizeB);
    if(hostB==NULL){
        
        printf("Host memory allocation is failed for hostB matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostC = (int*) malloc(sizeC);
    if(hostC==NULL){
        
        printf("Host memory allocation is failed for hostC matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    gold = (int*)malloc(sizeGold);
    if(gold==NULL){
        
        printf("Host memory allocation is failed for gold matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // printing matrix dimensions and sizes
    printf("The Dimensions of Matrix 'hostA' are : %d x %d \n", numARows, numACols);
    printf("The Dimensions of Matrix 'hostB' are : %d x %d \n", numBRows, numBCols);
    printf("The Dimensions of Matrix 'hostC' are : %d x %d \n", numCRows, numCCols);
    printf("The Dimensions of Matrix 'Gold' are : %d x %d \n", numGoldRows, numGoldCols);

    // fill source matrices
    initA(hostA, numARows, numACols);
    initB(hostB, numBRows, numBCols);

    // device memory allocation
    result = hipMalloc((void**) &deviceA, sizeA);
    if(result!=hipSuccess){
        printf("Device memory allocation is failed for deviceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void**)&deviceB, sizeB);
    if(result!=hipSuccess){
        printf("Device memory allocation is failed for deviceB matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void**)&deviceC, sizeC);
    if(result!=hipSuccess){
        printf("Device memory allocation is failed for deviceC matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // copy data from host matrices into device matrices
    result = hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
    if(result!=hipSuccess){
        printf("Host to device data copy is failed for deviceA matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);
    if(result!=hipSuccess){
        printf("Host to device data copy is failed for deviceB matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // CUDA kernel configuration
    dim3 dimGrid = (ceil((int)numBCols / (int)BLOCK_WIDTH), ceil((int)numARows/(int)BLOCK_WIDTH), 1);
    dim3 dimBlock = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // CUDA kernel for matrix multiplication
    
    matrixMultiplicationGPU <<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numACols, numBCols, numCCols);

    

    // copy data from device matrix into host matrix
    result = hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
    if(result != hipSuccess){
        printf("Device to Host data copy is failed for hostC matrix.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // matrix multiplication on host
    matrixMultiplicationCPU(hostA, hostB, gold, numARows, numACols, numBCols, numCCols);

    // comparison
    int breakValue = -1;
    bool bAccuracy = true;
    for(int i = 0; i < numCRows; i++){
        
        int val1 = gold[i];
        int val2 = hostC[i];
        if(val1 != val2){
            bAccuracy = false;
            breakValue = i;
            break;
        }
    }

    char str[128];
    if(bAccuracy == false){
        sprintf(str, "Comparison of CPU and GPU Matrix Multiplication is not accurate at array index %d", breakValue);
    }else{
        sprintf(str, "Comparison of CPU and GPU Matrix Multiplication is accurate.");
    }

    
    printf("%s\n", str);

    // cleanup
    cleanup();

    return(0);
}

void initA(int *data, int row, int col){

    int num=1;

    // code
    for(int i=0; i<row; i++){
        for(int j=0; j<col; j++){
            *(data + i * col + j) = num;
            num++;
        }
    }
}

void initB(int *data, int row, int col){

    int num = BLOCK_WIDTH;

    // code
    for(int i = 0; i < row; i++){
        for(int j = 0; j < col; j++){
            *(data + i * col + j) = num;
            num--;
        }
    }

}



void matrixMultiplicationCPU(int *A, int *B, int *C, int numARows,int numACols ,int numBCols, int numCCols){

    // code
    

    for(int i = 0 ; i < numARows; ++i){

        for(int j = 0; j < numBCols; ++j){

            int value = 1;
            for(int k = 0; k < numACols; ++k){

                int a = A[i * numACols + k];
                int b = B[k * numBCols + j];
                value += a * b;
            }
            C[i * numCCols + j] = value;
        }
    }
        
}

void cleanup(void){

    // code
    if(deviceC){
        hipFree(deviceC);
        deviceC = NULL;
    }

    if(deviceB){
        hipFree(deviceB);
        deviceB = NULL;

    }

    if(deviceA){
        hipFree(deviceA);
        deviceA = NULL;
    }

    if(gold){
        free(gold);
        gold = NULL;
    }

    if(hostC){
        free(hostC);
        hostC = NULL;
    }

    if(hostB){
        free(hostB);
        hostB = NULL;
    }

    if(hostA){
        free(hostA);
        hostA = NULL;
    }
}

